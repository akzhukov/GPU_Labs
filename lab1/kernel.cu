#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <algorithm>
#include <random>
#include <windows.h>

#define BLOCK_SIZE 16
#define MIN_VALUE -25
#define MAX_VALUE 25
#define MATRIX_SIZE 1000
#define DELTA 10e-10

using namespace std;


__global__ void matrixMult(double* __restrict__ A, double* __restrict__ B, double* __restrict__ C, size_t size)
{
	size_t i = blockDim.y * blockIdx.y + threadIdx.y;
	size_t j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= size || j >= size)
		return;

	size_t ind = i * size + j;
	C[ind] = 0;
	for (size_t k = 0; k < size; k++) {
		C[ind] += A[i * size + k] * B[k * size + j];
	}

}

float matrixMulOnGPU(double* A, double* B, double* C, size_t size) {

	double* dA, * dB, * dC;

	size_t numBytes = size * size * sizeof(double);

	hipEvent_t start, end;
	float time;

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks((size + threads.x - 1) / threads.x, (size + threads.y - 1) / threads.y);

	hipMalloc((void**)(&dA), numBytes);
	hipMalloc((void**)(&dB), numBytes);
	hipMalloc((void**)(&dC), numBytes);

	hipMemcpy(dA, A, numBytes, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(dB, B, numBytes, hipMemcpyKind::hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start, 0);

	matrixMult << <blocks, threads >> > (dA, dB, dC, size);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);

	hipEventDestroy(start);
	hipEventDestroy(end);

	hipMemcpy(C, dC, numBytes, hipMemcpyKind::hipMemcpyDeviceToHost);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	return time / 1000.0f;
}

float matrixMulOnCPU(double* A, double* B, double* C, size_t size) {

	LARGE_INTEGER frequency;
	QueryPerformanceFrequency(&frequency);

	LARGE_INTEGER start;
	QueryPerformanceCounter(&start);

	for (size_t i = 0; i < size; i++) {
		for (size_t j = 0; j < size; j++) {
			size_t ind = i * size + j;
			C[ind] = 0;
			for (size_t k = 0; k < size; k++) {
				C[ind] += A[i * size + k] * B[k * size + j];
			}
		}
	}

	LARGE_INTEGER end;
	QueryPerformanceCounter(&end);

	return (float)(end.QuadPart - start.QuadPart) / frequency.QuadPart;
}



bool compareMatrices(double* A, double* B, size_t size) {
	size_t numCells = size * size;
	for (int i = 0; i < numCells; i++) {
		if (std::abs(A[i] - B[i]) > DELTA) {
			return false;
		}
	}
	return true;
}

double* generateRandomMatrix(size_t size)
{
	random_device rd;
	mt19937 gen(rd());
	uniform_int_distribution<> distr(MIN_VALUE, MAX_VALUE);

	size_t numElements = size * size;
	double* matrix = new double[numElements];

	for (size_t i = 0; i < numElements; i++) {
		matrix[i] = distr(gen);
	}

	return matrix;
}

int main()
{
	setlocale(LC_ALL, "Russian");
	double* hA = generateRandomMatrix(MATRIX_SIZE);
	double* hB = generateRandomMatrix(MATRIX_SIZE);
	double* hCGPU = new double[MATRIX_SIZE * MATRIX_SIZE];
	double* hCCPU = new double[MATRIX_SIZE * MATRIX_SIZE];
	double time;

	cout << "Начато вычисление на GPU..." << endl;
	time = matrixMulOnGPU(hA, hB, hCGPU, MATRIX_SIZE);
	cout << "Вычисление на GPU завершено за " << time << "секунд" << endl;
	cout << "Начато вычисление на CPU..." << endl;
	time = matrixMulOnCPU(hA, hB, hCCPU, MATRIX_SIZE);
	cout << "Вычисление на CPU завершено за " << time << "секунд" << endl;
	cout << "Сравнение матриц:" << endl;
	if (compareMatrices(hCGPU, hCCPU, MATRIX_SIZE))
		cout << "Результаты вычислений совпадают!" << endl;
	else
		cout << "Результаты вычислений НЕ совпадают!" << endl;

	delete[] hA;
	delete[] hB;
	delete[] hCGPU;
	delete[] hCCPU;
	return 0;
}
